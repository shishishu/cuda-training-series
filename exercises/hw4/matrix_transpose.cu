/*
src code: https://github.com/NVIDIA-developer-blog/code-samples/blob/master/series/cuda-cpp/transpose/transpose.cu
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

void postprocess(const float *ref, const float *res, int n, float ms)
{
  bool passed = true;
  for (int i = 0; i < n; i++) {
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  }
  if (passed)
    printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms);
}

// simple copy kernel
// Used as reference case representing best effective bandwidth.
__global__ void copy(float *odata, const float *idata) {
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;  // be careful, not BLOCK_ROWS
  int width = gridDim.x * TILE_DIM;
  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[(y + j) * width + x] = idata[(y + j) * width + x];
  }
}

// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(float *odata, const float *idata) {
  __shared__ float tile[TILE_DIM * TILE_DIM];
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    tile[(threadIdx.y + j) * TILE_DIM + threadIdx.x] = idata[(y + j) * width + x];
  }
  __syncthreads();
  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[(y + j) * width + x] = tile[(threadIdx.y + j) * TILE_DIM + threadIdx.x];
  }
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive(float *odata, const float *idata) {
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
    odata[x * width + (y + j)] = idata[(y + j) * width + x]; 
  }
}

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)                                                                                                            
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded 
// to avoid shared memory bank conflicts.
__global__ void transposeCoalesced2(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

int main(){
  const int nx = 1024;
  const int ny = 1024;
  const int mem_size = nx * ny * sizeof(float);

  dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
  printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n", nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
  printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n", dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

  float *h_idata = (float*)malloc(mem_size);
  float *h_cdata = (float*)malloc(mem_size);
  float *h_tdata = (float*)malloc(mem_size);
  float *gold = (float*)malloc(mem_size);

  float *d_idata, *d_cdata, *d_tdata;
  hipMalloc(&d_idata, mem_size);
  hipMalloc(&d_cdata, mem_size);
  hipMalloc(&d_tdata, mem_size);

  for (int j = 0; j < ny; j++) {
    for (int i = 0; i < nx; i++) {
      h_idata[j * nx + i ] = j * nx + i;
    }
  }

  for (int j = 0; j < ny; j++) {
    for (int i = 0; i < nx; i++) {
      gold[j * nx + i ] = h_idata[i * nx + j];
    }
  }

  hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice);

  // timing
  hipEvent_t startEvent, stopEvent;
  hipEventCreate(&startEvent);
  hipEventCreate(&stopEvent);
  float ms;
  
  hipError_t err;
  printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");

  // copy
  printf("%25s", "copy");
  checkCuda(hipMemset(d_cdata, 0, mem_size));
  // warmup
  copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    exit(1);
  }
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < NUM_REPS; i++) {
    copy<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost));
  postprocess(h_idata, h_cdata, nx*ny, ms);

  // copySharedMem
  printf("%25s", "shared memory copy");
  checkCuda(hipMemset(d_cdata, 0, mem_size));
  // warmup
  copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    exit(1);
  }
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < NUM_REPS; i++) {
    copySharedMem<<<dimGrid, dimBlock>>>(d_cdata, d_idata);
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  checkCuda(hipMemcpy(h_cdata, d_cdata, mem_size, hipMemcpyDeviceToHost));
  postprocess(h_idata, h_cdata, nx*ny, ms);

  // transposeNaive
  printf("%25s", "naive transpose");
  checkCuda(hipMemset(d_tdata, 0, mem_size));
  // warmup
  transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    exit(1);
  }
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < NUM_REPS; i++) {
    transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
  postprocess(gold, h_tdata, nx*ny, ms);

  // transposeCoalesced
  printf("%25s", "coalesced transpose");
  checkCuda(hipMemset(d_tdata, 0, mem_size));
  // warmup
  transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    exit(1);
  }
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < NUM_REPS; i++) {
    transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
  postprocess(gold, h_tdata, nx*ny, ms);

  // transposeCoalesced2
  printf("%25s", "coalesced transpose 2");
  checkCuda(hipMemset(d_tdata, 0, mem_size));
  // warmup
  transposeCoalesced2<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    exit(1);
  }
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < NUM_REPS; i++) {
    transposeCoalesced2<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  checkCuda(hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost));
  postprocess(gold, h_tdata, nx*ny, ms);

  return 0;
}
