# matrix_sums.cu
```
nvprof ./matrix_sums
```
```
==822815== NVPROF is profiling process 822815, command: ./matrix_sums
row sums correct!
column sums correct!
==822815== Profiling application: ./matrix_sums
==822815== Profiling result:
            Type  Time(%)      Time     Calls       Avg       Min       Max  Name
 GPU activities:   97.39%  225.83ms         1  225.83ms  225.83ms  225.83ms  [CUDA memcpy HtoD]
                    1.48%  3.4310ms         1  3.4310ms  3.4310ms  3.4310ms  row_sums(float const *, float*, unsigned long)
                    1.12%  2.6016ms         1  2.6016ms  2.6016ms  2.6016ms  column_sums(float const *, float*, unsigned long)
                    0.01%  12.639us         2  6.3190us  6.1750us  6.4640us  [CUDA memcpy DtoH]
                    0.00%  3.2640us         1  3.2640us  3.2640us  3.2640us  [CUDA memset]
      API calls:   62.19%  232.14ms         3  77.382ms  2.6272ms  226.00ms  hipMemcpy
                   37.54%  140.15ms         2  70.073ms  123.07us  140.02ms  hipMalloc
                    0.22%  830.98us       404  2.0560us     148ns  91.088us  hipDeviceGetAttribute
                    0.02%  81.781us         4  20.445us  17.911us  27.136us  hipDeviceGetName
                    0.01%  42.525us         2  21.262us  8.5890us  33.936us  hipLaunchKernel
                    0.00%  15.098us         1  15.098us  15.098us  15.098us  hipMemset
                    0.00%  10.641us         4  2.6600us     852ns  7.2090us  hipDeviceGetPCIBusId
                    0.00%  1.8140us         8     226ns     140ns     522ns  hipDeviceGet
                    0.00%  1.5530us         6     258ns     156ns     626ns  hipGetLastError
                    0.00%  1.0730us         4     268ns     190ns     437ns  hipDeviceTotalMem
                    0.00%  1.0200us         3     340ns     181ns     656ns  hipGetDeviceCount
                    0.00%     906ns         4     226ns     196ns     265ns  hipDeviceGetUuid
                    0.00%     362ns         1     362ns     362ns     362ns  cuModuleGetLoadingMode

```

```
ncu ./matrix_sums
```
```
==PROF== Connected to process 823622 (/data/docker/zhouxiangjun/cuda-training-series/exercises/hw4/matrix_sums)
==PROF== Profiling "row_sums" - 0: 0%....50%....100% - 19 passes
row sums correct!
==PROF== Profiling "column_sums" - 1: 0%....50%....100% - 19 passes
column sums correct!
==PROF== Disconnected from process 823622
[823622] matrix_sums@127.0.0.1
  row_sums(const float *, float *, unsigned long), 2025-Apr-22 00:30:24, Context 1, Stream 7
    Section: GPU Speed Of Light Throughput
    ---------------------------------------------------------------------- --------------- ------------------------------
    DRAM Frequency                                                           cycle/usecond                         872.95
    SM Frequency                                                             cycle/nsecond                           1.29
    Elapsed Cycles                                                                   cycle                      4,412,358
    Memory [%]                                                                           %                          76.17
    DRAM Throughput                                                                      %                          35.14
    Duration                                                                       msecond                           3.42
    L1/TEX Cache Throughput                                                              %                          95.94
    L2 Cache Throughput                                                                  %                          12.57
    SM Active Cycles                                                                 cycle                   3,497,634.02
    Compute (SM) [%]                                                                     %                           2.68
    ---------------------------------------------------------------------- --------------- ------------------------------
    WRN   Memory is more heavily utilized than Compute: Look at the Memory Workload Analysis section to identify the L1 
          bottleneck. Check memory replay (coalescing) metrics to make sure you're efficiently utilizing the bytes      
          transferred. Also consider whether it is possible to do more work per memory access (kernel fusion) or        
          whether there are values you can (re)compute.                                                                 

    Section: Launch Statistics
    ---------------------------------------------------------------------- --------------- ------------------------------
    Block Size                                                                                                        256
    Function Cache Configuration                                                                  hipFuncCachePreferNone
    Grid Size                                                                                                          64
    Registers Per Thread                                                   register/thread                             21
    Shared Memory Configuration Size                                                  byte                              0
    Driver Shared Memory Per Block                                              byte/block                              0
    Dynamic Shared Memory Per Block                                             byte/block                              0
    Static Shared Memory Per Block                                              byte/block                              0
    Threads                                                                         thread                         16,384
    Waves Per SM                                                                                                     0.10
    ---------------------------------------------------------------------- --------------- ------------------------------
    WRN   The grid for this launch is configured to execute only 64 blocks, which is less than the GPU's 80             
          multiprocessors. This can underutilize some multiprocessors. If you do not intend to execute this kernel      
          concurrently with other workloads, consider reducing the block size to have at least one block per            
          multiprocessor or increase the size of the grid to fully utilize the available hardware resources. See the    
          Hardware Model (https://docs.nvidia.com/nsight-compute/ProfilingGuide/index.html#metrics-hw-model)            
          description for more details on launch configurations.                                                        

    Section: Occupancy
    ---------------------------------------------------------------------- --------------- ------------------------------
    Block Limit SM                                                                   block                             32
    Block Limit Registers                                                            block                             10
    Block Limit Shared Mem                                                           block                             32
    Block Limit Warps                                                                block                              8
    Theoretical Active Warps per SM                                                   warp                             64
    Theoretical Occupancy                                                                %                            100
    Achieved Occupancy                                                                   %                          12.50
    Achieved Active Warps Per SM                                                      warp                           8.00
    ---------------------------------------------------------------------- --------------- ------------------------------
    WRN   This kernel's theoretical occupancy is not impacted by any block limit. The difference between calculated     
          theoretical (100.0%) and measured achieved occupancy (12.5%) can be the result of warp scheduling overheads   
          or workload imbalances during the kernel execution. Load imbalances can occur between warps within a block    
          as well as across blocks of the same kernel. See the CUDA Best Practices Guide                                
          (https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#occupancy) for more details on           
          optimizing occupancy.                                                                                         

  column_sums(const float *, float *, unsigned long), 2025-Apr-22 00:30:24, Context 1, Stream 7
    Section: GPU Speed Of Light Throughput
    ---------------------------------------------------------------------- --------------- ------------------------------
    DRAM Frequency                                                           cycle/usecond                         879.08
    SM Frequency                                                             cycle/nsecond                           1.30
    Elapsed Cycles                                                                   cycle                      3,307,618
    Memory [%]                                                                           %                          46.87
    DRAM Throughput                                                                      %                          46.87
    Duration                                                                       msecond                           2.54
    L1/TEX Cache Throughput                                                              %                          16.47
    L2 Cache Throughput                                                                  %                          16.76
    SM Active Cycles                                                                 cycle                   2,546,831.92
    Compute (SM) [%]                                                                     %                           4.57
    ---------------------------------------------------------------------- --------------- ------------------------------
    WRN   This kernel grid is too small to fill the available resources on this device, resulting in only 0.1 full      
          waves across all SMs. Look at Launch Statistics for more details.                                             

    Section: Launch Statistics
    ---------------------------------------------------------------------- --------------- ------------------------------
    Block Size                                                                                                        256
    Function Cache Configuration                                                                  hipFuncCachePreferNone
    Grid Size                                                                                                          64
    Registers Per Thread                                                   register/thread                             26
    Shared Memory Configuration Size                                                  byte                              0
    Driver Shared Memory Per Block                                              byte/block                              0
    Dynamic Shared Memory Per Block                                             byte/block                              0
    Static Shared Memory Per Block                                              byte/block                              0
    Threads                                                                         thread                         16,384
    Waves Per SM                                                                                                     0.10
    ---------------------------------------------------------------------- --------------- ------------------------------
    WRN   The grid for this launch is configured to execute only 64 blocks, which is less than the GPU's 80             
          multiprocessors. This can underutilize some multiprocessors. If you do not intend to execute this kernel      
          concurrently with other workloads, consider reducing the block size to have at least one block per            
          multiprocessor or increase the size of the grid to fully utilize the available hardware resources. See the    
          Hardware Model (https://docs.nvidia.com/nsight-compute/ProfilingGuide/index.html#metrics-hw-model)            
          description for more details on launch configurations.                                                        

    Section: Occupancy
    ---------------------------------------------------------------------- --------------- ------------------------------
    Block Limit SM                                                                   block                             32
    Block Limit Registers                                                            block                              8
    Block Limit Shared Mem                                                           block                             32
    Block Limit Warps                                                                block                              8
    Theoretical Active Warps per SM                                                   warp                             64
    Theoretical Occupancy                                                                %                            100
    Achieved Occupancy                                                                   %                          12.45
    Achieved Active Warps Per SM                                                      warp                           7.97
    ---------------------------------------------------------------------- --------------- ------------------------------
    WRN   This kernel's theoretical occupancy is not impacted by any block limit. The difference between calculated     
          theoretical (100.0%) and measured achieved occupancy (12.5%) can be the result of warp scheduling overheads   
          or workload imbalances during the kernel execution. Load imbalances can occur between warps within a block    
          as well as across blocks of the same kernel. See the CUDA Best Practices Guide                                
          (https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#occupancy) for more details on           
          optimizing occupancy.        
```

```
ncu --metrics l1tex__t_sectors_pipe_lsu_mem_global_op_ld.sum,l1tex__t_requests_pipe_lsu_mem_global_op_ld.sum ./matrix_sums
```
```
==PROF== Connected to process 824524 (/data/docker/zhouxiangjun/cuda-training-series/exercises/hw4/matrix_sums)
==PROF== Profiling "row_sums" - 0: 0%....50%....100% - 4 passes
row sums correct!
==PROF== Profiling "column_sums" - 1: 0%....50%....100% - 4 passes
column sums correct!
==PROF== Disconnected from process 824524
[824524] matrix_sums@127.0.0.1
  row_sums(const float *, float *, unsigned long), 2025-Apr-22 00:31:34, Context 1, Stream 7
    Section: Command line profiler metrics
    ---------------------------------------------------------------------- --------------- ------------------------------
    l1tex__t_requests_pipe_lsu_mem_global_op_ld.sum                                request                      8,388,608
    l1tex__t_sectors_pipe_lsu_mem_global_op_ld.sum                                  sector                    268,435,456
    ---------------------------------------------------------------------- --------------- ------------------------------

  column_sums(const float *, float *, unsigned long), 2025-Apr-22 00:31:34, Context 1, Stream 7
    Section: Command line profiler metrics
    ---------------------------------------------------------------------- --------------- ------------------------------
    l1tex__t_requests_pipe_lsu_mem_global_op_ld.sum                                request                      8,388,608
    l1tex__t_sectors_pipe_lsu_mem_global_op_ld.sum                                  sector                     33,554,432
    ---------------------------------------------------------------------- --------------- ------------------------------

```
