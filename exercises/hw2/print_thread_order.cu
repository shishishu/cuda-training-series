
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_thread_order() {
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    printf("Block(%d,%d): threadIdx=(%d,%d), linear_id=%d\n",
           blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, tid);
}

int main() {
    dim3 block(32, 2); // 32x2的Block
    dim3 grid(1, 1);
    print_thread_order<<<grid, block>>>();
    hipDeviceSynchronize();
    return 0;
}
